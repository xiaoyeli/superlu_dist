#include "hip/hip_runtime.h"
#include "superlu_ddefs.h"
#include "lupanels_GPU.cuh"
#include "lupanels.hpp"

hipError_t checkCudaLocal(hipError_t result)
{
    // #if defined(DEBUG) || defined(_DEBUG)
    // printf("Checking cuda\n");
    if (result != hipSuccess)
    {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    // #endif
    return result;
}

__global__ void indirectCopy(double *dest, double *src, int_t *idx, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        dest[idx[i]] = src[i];
}

/**
 * @file schurCompUpdate.cu
 * @brief This function copies the packed buffers to GPU and performs the sparse
   initialization on GPU call indirectCopy, this is the kernel
 * @param gpuValBasePtr is the base pointer of the GPU matrix
 * @param valBufferPacked is the packed buffer of the matrix
 * @param valIdx is the index of the packed buffer
 */
void copyToGPU(double *gpuValBasePtr, std::vector<double> &valBufferPacked,
               std::vector<int_t> &valIdx)
{
    int nnzCount = valBufferPacked.size();
    // calculate the size of the packed buffers
    int_t gpuLvalSizePacked = nnzCount * sizeof(double);
    int_t gpuLidxSizePacked = nnzCount * sizeof(int_t);
    // allocate the memory for the packed buffers on GPU
    double *dlvalPacked;
    int_t *dlidxPacked;
    hipMalloc(&dlvalPacked, gpuLvalSizePacked);
    hipMalloc(&dlidxPacked, gpuLidxSizePacked);
    // copy the packed buffers from CPU to GPU
    hipMemcpy(dlvalPacked, valBufferPacked.data(), gpuLvalSizePacked, hipMemcpyHostToDevice);
    hipMemcpy(dlidxPacked, valIdx.data(), gpuLidxSizePacked, hipMemcpyHostToDevice);
    // perform the sparse initialization on GPU call indirectCopy
    const int ThreadblockSize = 256;
    int nThreadBlocks = (nnzCount + ThreadblockSize - 1) / ThreadblockSize;
    indirectCopy<<<nThreadBlocks, ThreadblockSize>>>(
        gpuValBasePtr, dlvalPacked, dlidxPacked, nnzCount);
    // wait for it to finish and free dlvalPacked and dlidxPacked
    hipDeviceSynchronize();
    hipFree(dlvalPacked);
    hipFree(dlidxPacked);
}

// copy the panel to GPU
void copyToGPU_Sparse(double *gpuValBasePtr, double *valBuffer, int_t gpuLvalSize)
{
    // sparse Initialization for GPU, this is the experimental code
    // find non-zero elements in the panel, their location and values  and copy to GPU
    int numDoubles = gpuLvalSize / sizeof(double);
    std::vector<double> valBufferPacked;
    std::vector<int_t> valIdx;
    for (int_t i = 0; i < numDoubles; i++)
    {
        if (valBuffer[i] != 0)
        {
            valBufferPacked.push_back(valBuffer[i]);
            valIdx.push_back(i);
        }
    }
    printf("%d non-zero elements in the panel, wrt original=%d\n", valBufferPacked.size(), numDoubles);
    // get the size of the packed buffers and allocate memory on GPU
    copyToGPU(gpuValBasePtr, valBufferPacked, valIdx);
}

#define NDEBUG
__device__
    int_t
    lpanelGPU_t::find(int_t k)
{
    int threadId = threadIdx.x;
    __shared__ int idx;
    __shared__ int found;
    if (!threadId)
    {
        idx = -1;
        found = 0;
    }

    int nThreads = blockDim.x;
    int blocksPerThreads = CEILING(nblocks(), nThreads);
    __syncthreads();
    for (int blk = blocksPerThreads * threadIdx.x;
         blk < blocksPerThreads * (threadIdx.x + 1);
         blk++)
    {
        // if(found) break;

        if (blk < nblocks())
        {
            if (k == gid(blk))
            {
                idx = blk;
                found = 1;
            }
        }
    }
    __syncthreads();
    return idx;
}

__device__
    int_t
    upanelGPU_t::find(int_t k)
{
    int threadId = threadIdx.x;
    __shared__ int idx;
    __shared__ int found;
    if (!threadId)
    {
        idx = -1;
        found = 0;
    }
    __syncthreads();

    int nThreads = blockDim.x;
    int blocksPerThreads = CEILING(nblocks(), nThreads);

    for (int blk = blocksPerThreads * threadIdx.x;
         blk < blocksPerThreads * (threadIdx.x + 1);
         blk++)
    {
        // if(found) break;

        if (blk < nblocks())
        {
            if (k == gid(blk))
            {
                idx = blk;
                found = 1;
            }
        }
    }
    __syncthreads();
    return idx;
}

__device__ int computeIndirectMapGPU(int *rcS2D, int_t srcLen, int_t *srcVec,
                                     int_t dstLen, int_t *dstVec,
                                     int *dstIdx)
{
    int threadId = threadIdx.x;
    if (dstVec == NULL) /*uncompressed dimension*/
    {
        if (threadId < srcLen)
            rcS2D[threadId] = srcVec[threadId];
        __syncthreads();
        return 0;
    }

    if (threadId < dstLen)
        dstIdx[dstVec[threadId]] = threadId;
    __syncthreads();

    if (threadId < srcLen)
        rcS2D[threadId] = dstIdx[srcVec[threadId]];
    __syncthreads();

    return 0;
}

__global__ void scatterGPU(
    int iSt, int jSt,
    double *gemmBuff, int LDgemmBuff,
    lpanelGPU_t lpanel, upanelGPU_t upanel,
    LUstructGPU_t *dA)
{

    // calculate gi,gj
    int ii = iSt + blockIdx.x;
    int jj = jSt + blockIdx.y;
    int threadId = threadIdx.x;

    int gi = lpanel.gid(ii);
    int gj = upanel.gid(jj);
#ifndef NDEBUG
    if (!threadId)
        printf("Scattering to (%d, %d) \n", gi, gj);
#endif
    double *Dst;
    int_t lddst;
    int_t dstRowLen, dstColLen;
    int_t *dstRowList;
    int_t *dstColList;
    int li, lj;
    if (gj > gi) // its in upanel
    {
        li = dA->g2lRow(gi);
        lj = dA->uPanelVec[li].find(gj);
        Dst = dA->uPanelVec[li].blkPtr(lj);
        lddst = dA->supersize(gi);
        dstRowLen = dA->supersize(gi);
        dstRowList = NULL;
        dstColLen = dA->uPanelVec[li].nbcol(lj);
        dstColList = dA->uPanelVec[li].colList(lj);
    }
    else
    {
        lj = dA->g2lCol(gj);
        li = dA->lPanelVec[lj].find(gi);
        Dst = dA->lPanelVec[lj].blkPtr(li);
        lddst = dA->lPanelVec[lj].LDA();
        dstRowLen = dA->lPanelVec[lj].nbrow(li);
        dstRowList = dA->lPanelVec[lj].rowList(li);
        // if(!threadId )
        // printf("Scattering to (%d, %d) by %d li=%d\n",gi, gj,threadId,li);
        dstColLen = dA->supersize(gj);
        dstColList = NULL;
    }

    // compute source row to dest row mapping
    int maxSuperSize = dA->maxSuperSize;
    extern __shared__ int baseSharedPtr[];
    int *rowS2D = baseSharedPtr;
    int *colS2D = &rowS2D[maxSuperSize];
    int *dstIdx = &colS2D[maxSuperSize];

    int nrows = lpanel.nbrow(ii);
    int ncols = upanel.nbcol(jj);
    // lpanel.rowList(ii), upanel.colList(jj)

    computeIndirectMapGPU(rowS2D, nrows, lpanel.rowList(ii),
                          dstRowLen, dstRowList, dstIdx);

    // compute source col to dest col mapping
    computeIndirectMapGPU(colS2D, ncols, upanel.colList(jj),
                          dstColLen, dstColList, dstIdx);

    int nThreads = blockDim.x;
    int colsPerThreadBlock = nThreads / nrows;

    int rowOff = lpanel.stRow(ii) - lpanel.stRow(iSt);
    int colOff = upanel.stCol(jj) - upanel.stCol(jSt);
    double *Src = &gemmBuff[rowOff + colOff * LDgemmBuff];
    int ldsrc = LDgemmBuff;
    // TODO: this seems inefficient
    if (threadId < nrows * colsPerThreadBlock)
    {
        /* 1D threads are logically arranged in 2D shape. */
        int i = threadId % nrows;
        int j = threadId / nrows;

#pragma unroll 4
        while (j < ncols)
        {

#define ATOMIC_SCATTER
// Atomic Scatter is need if I want to perform multiple Schur Complement
//  update concurrently
#ifdef ATOMIC_SCATTER
            atomicAdd(&Dst[rowS2D[i] + lddst * colS2D[j]], -Src[i + ldsrc * j]);
#else
            Dst[rowS2D[i] + lddst * colS2D[j]] -= Src[i + ldsrc * j];
#endif
            j += colsPerThreadBlock;
        }
    }

    __syncthreads();
}

int_t LUstruct_v100::dSchurComplementUpdateGPU(
    int streamId,
    int_t k, lpanel_t &lpanel, upanel_t &upanel)
{

    if (lpanel.isEmpty() || upanel.isEmpty())
        return 0;

    int_t st_lb = 0;
    if (myrow == krow(k))
        st_lb = 1;

    int_t nlb = lpanel.nblocks();
    int_t nub = upanel.nblocks();

    int iSt = st_lb;
    int iEnd = iSt;

    int nrows = lpanel.stRow(nlb) - lpanel.stRow(st_lb);
    int ncols = upanel.nzcols();

    int maxGemmRows = nrows;
    int maxGemmCols = ncols;
    // entire gemm doesn't fit in gemm buffer
    if (nrows * ncols > A_gpu.gemmBufferSize)
    {
        int maxGemmOpSize = (int)sqrt(A_gpu.gemmBufferSize);
        int numberofRowChunks = (nrows + maxGemmOpSize - 1) / maxGemmOpSize;
        maxGemmRows = nrows / numberofRowChunks;
        maxGemmCols = A_gpu.gemmBufferSize / maxGemmRows;
    }

    while (iEnd < nlb)
    {
        iSt = iEnd;
        iEnd = lpanel.getEndBlock(iSt, maxGemmRows);

        assert(iEnd > iSt);
        int jSt = 0;
        int jEnd = 0;
        while (jEnd < nub)
        {
            jSt = jEnd;
            jEnd = upanel.getEndBlock(jSt, maxGemmCols);
            assert(jEnd > jSt);
            hipblasHandle_t handle = A_gpu.cuHandles[streamId];
            hipStream_t cuStream = A_gpu.cuStreams[streamId];
            hipblasSetStream(handle, cuStream);
            int gemm_m = lpanel.stRow(iEnd) - lpanel.stRow(iSt);
            int gemm_n = upanel.stCol(jEnd) - upanel.stCol(jSt);
            int gemm_k = supersize(k);
            double alpha = 1.0;
            double beta = 0.0;
#ifndef NDEBUG
            printf("m=%d, n=%d, k=%d\n", gemm_m, gemm_n, gemm_k);
#endif
            hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                        gemm_m, gemm_n, gemm_k, &alpha,
                        lpanel.blkPtrGPU(iSt), lpanel.LDA(),
                        upanel.blkPtrGPU(jSt), upanel.LDA(), &beta,
                        A_gpu.gpuGemmBuffs[streamId], gemm_m);

            // setting up scatter
            dim3 dimBlock(ldt); // 1d thread
            dim3 dimGrid(iEnd - iSt, jEnd - jSt);
            size_t sharedMemorySize = 3 * A_gpu.maxSuperSize * sizeof(int_t);

            scatterGPU<<<dimGrid, dimBlock, sharedMemorySize, cuStream>>>(
                iSt, jSt,
                A_gpu.gpuGemmBuffs[streamId], gemm_m,
                lpanel.gpuPanel, upanel.gpuPanel, dA_gpu);
        }
    }
    checkCudaLocal(hipStreamSynchronize(A_gpu.cuStreams[streamId]));
    return 0;
}

int_t LUstruct_v100::lookAheadUpdateGPU(
    int streamId,
    int_t k, int_t laIdx, lpanel_t &lpanel, upanel_t &upanel)
{
    if (lpanel.isEmpty() || upanel.isEmpty())
        return 0;

    int_t st_lb = 0;
    if (myrow == krow(k))
        st_lb = 1;

    int_t nlb = lpanel.nblocks();
    int_t nub = upanel.nblocks();

    int_t laILoc = lpanel.find(laIdx);
    int_t laJLoc = upanel.find(laIdx);

    int iSt = st_lb;
    int jSt = 0;

    /* call look ahead update on Lpanel*/
    if (laJLoc != GLOBAL_BLOCK_NOT_FOUND)
        dSchurCompUpdatePartGPU(
            iSt, nlb, laJLoc, laJLoc + 1,
            k, lpanel, upanel,
            A_gpu.lookAheadLHandle[streamId], A_gpu.lookAheadLStream[streamId],
            A_gpu.lookAheadLGemmBuffer[streamId]);

    /* call look ahead update on Upanel*/
    if (laILoc != GLOBAL_BLOCK_NOT_FOUND)
    {
        dSchurCompUpdatePartGPU(
            laILoc, laILoc + 1, jSt, laJLoc,
            k, lpanel, upanel,
            A_gpu.lookAheadUHandle[streamId], A_gpu.lookAheadUStream[streamId],
            A_gpu.lookAheadUGemmBuffer[streamId]);
        dSchurCompUpdatePartGPU(
            laILoc, laILoc + 1, laJLoc + 1, nub,
            k, lpanel, upanel,
            A_gpu.lookAheadUHandle[streamId], A_gpu.lookAheadUStream[streamId],
            A_gpu.lookAheadUGemmBuffer[streamId]);
    }

    // checkCudaLocal(hipStreamSynchronize(A_gpu.lookAheadLStream[streamId]));
    // checkCudaLocal(hipStreamSynchronize(A_gpu.lookAheadUStream[streamId]));

    return 0;
}

int_t LUstruct_v100::SyncLookAheadUpdate(int streamId)
{
    checkCudaLocal(hipStreamSynchronize(A_gpu.lookAheadLStream[streamId]));
    checkCudaLocal(hipStreamSynchronize(A_gpu.lookAheadUStream[streamId]));

    return 0;
}

int_t LUstruct_v100::dSchurCompUpdateExcludeOneGPU(
    int streamId,
    int_t k, int_t ex, // suypernodes to be excluded
    lpanel_t &lpanel, upanel_t &upanel)
{
    if (lpanel.isEmpty() || upanel.isEmpty())
        return 0;

    int_t st_lb = 0;
    if (myrow == krow(k))
        st_lb = 1;

    int_t nlb = lpanel.nblocks();
    int_t nub = upanel.nblocks();

    int_t exILoc = lpanel.find(ex);
    int_t exJLoc = upanel.find(ex);

    dSchurCompUpLimitedMem(
        streamId,
        st_lb, exILoc, 0, exJLoc,
        k, lpanel, upanel);

    dSchurCompUpLimitedMem(
        streamId,
        st_lb, exILoc, exJLoc + 1, nub,
        k, lpanel, upanel);

    int_t nextStI = exILoc + 1;
    if (exILoc == GLOBAL_BLOCK_NOT_FOUND)
        nextStI = st_lb;
    /*
    for j we don't need to change since, if exJLoc == GLOBAL_BLOCK_NOT_FOUND =-1
    then exJLoc+1 =0 will work out correctly as starting j
    */
    dSchurCompUpLimitedMem(
        streamId,
        nextStI, nlb, 0, exJLoc,
        k, lpanel, upanel);

    dSchurCompUpLimitedMem(
        streamId,
        nextStI, nlb, exJLoc + 1, nub,
        k, lpanel, upanel);

    // checkCudaLocal(hipStreamSynchronize(A_gpu.cuStreams[streamId]));
    return 0;
}

int_t LUstruct_v100::dSchurCompUpdatePartGPU(
    int_t iSt, int_t iEnd, int_t jSt, int_t jEnd,
    int_t k, lpanel_t &lpanel, upanel_t &upanel,
    hipblasHandle_t handle, hipStream_t cuStream,
    double *gemmBuff)
{
    if (iSt >= iEnd || jSt >= jEnd)
        return 0;

    hipblasSetStream(handle, cuStream);
    int gemm_m = lpanel.stRow(iEnd) - lpanel.stRow(iSt);
    int gemm_n = upanel.stCol(jEnd) - upanel.stCol(jSt);
    int gemm_k = supersize(k);
    double alpha = 1.0;
    double beta = 0.0;
#ifndef NDEBUG
    printf("m=%d, n=%d, k=%d\n", gemm_m, gemm_n, gemm_k);
#endif
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                gemm_m, gemm_n, gemm_k, &alpha,
                lpanel.blkPtrGPU(iSt), lpanel.LDA(),
                upanel.blkPtrGPU(jSt), upanel.LDA(), &beta,
                gemmBuff, gemm_m);

    // setting up scatter
    dim3 dimBlock(ldt); // 1d thread
    dim3 dimGrid(iEnd - iSt, jEnd - jSt);
    size_t sharedMemorySize = 3 * A_gpu.maxSuperSize * sizeof(int_t);

    scatterGPU<<<dimGrid, dimBlock, sharedMemorySize, cuStream>>>(
        iSt, jSt,
        gemmBuff, gemm_m,
        lpanel.gpuPanel, upanel.gpuPanel, dA_gpu);

    return 0;
}

int_t LUstruct_v100::dSchurCompUpLimitedMem(
    int streamId,
    int_t lStart, int_t lEnd,
    int_t uStart, int_t uEnd,
    int_t k, lpanel_t &lpanel, upanel_t &upanel)
{

    if (lStart >= lEnd || uStart >= uEnd)
        return 0;
    int iSt = lStart;
    int iEnd = iSt;
    int nrows = lpanel.stRow(lEnd) - lpanel.stRow(lStart);
    int ncols = upanel.stCol(uEnd) - upanel.stCol(uStart);

    int maxGemmRows = nrows;
    int maxGemmCols = ncols;
    // entire gemm doesn't fit in gemm buffer
    if (nrows * ncols > A_gpu.gemmBufferSize)
    {
        int maxGemmOpSize = (int)sqrt(A_gpu.gemmBufferSize);
        int numberofRowChunks = (nrows + maxGemmOpSize - 1) / maxGemmOpSize;
        maxGemmRows = nrows / numberofRowChunks;
        maxGemmCols = A_gpu.gemmBufferSize / maxGemmRows;
    }

    while (iEnd < lEnd)
    {
        iSt = iEnd;
        iEnd = lpanel.getEndBlock(iSt, maxGemmRows);
        if (iEnd > lEnd)
            iEnd = lEnd;

        assert(iEnd > iSt);
        int jSt = uStart;
        int jEnd = uStart;
        while (jEnd < uEnd)
        {
            jSt = jEnd;
            jEnd = upanel.getEndBlock(jSt, maxGemmCols);
            if (jEnd > uEnd)
                jEnd = uEnd;

            hipblasHandle_t handle = A_gpu.cuHandles[streamId];
            hipStream_t cuStream = A_gpu.cuStreams[streamId];
            dSchurCompUpdatePartGPU(iSt, iEnd, jSt, jEnd,
                                    k, lpanel, upanel, handle, cuStream, A_gpu.gpuGemmBuffs[streamId]);
        }
    }

    return 0;
}

int getMPIProcsPerGPU()
{
    if (!(getenv("MPI_PROCESS_PER_GPU")))
    {
        return 1;
    }
    else
    {
        int devCount;
        hipGetDeviceCount(&devCount);
        int envCount = atoi(getenv("MPI_PROCESS_PER_GPU"));
        envCount = SUPERLU_MAX(envCount, 0);
        return SUPERLU_MIN(envCount, devCount);
    }
}

#define USABLE_GPU_MEM_FRACTION 0.9

size_t getGPUMemPerProcs(MPI_Comm baseCommunicator)
{

    size_t mfree, mtotal;
    // TODO: shared memory communicator should be part of
    //  LU struct
    //  MPI_Comm sharedComm;
    //  MPI_Comm_split_type(baseCommunicator, MPI_COMM_TYPE_SHARED,
    //                      0, MPI_INFO_NULL, &sharedComm);
    //  MPI_Barrier(sharedComm);
    hipMemGetInfo(&mfree, &mtotal);
    // MPI_Barrier(sharedComm);
    // MPI_Comm_free(&sharedComm);
#if 0
    printf("Total memory %zu & free memory %zu\n", mtotal, mfree);
#endif
    return (size_t)(USABLE_GPU_MEM_FRACTION * (double)mfree) / getMPIProcsPerGPU();
}

int_t LUstruct_v100::setLUstruct_GPU()
{

    A_gpu.Pr = Pr;
    A_gpu.Pc = Pc;
    A_gpu.maxSuperSize = ldt;

    /*Mapping to device*/
    int deviceCount;
    hipGetDeviceCount(&deviceCount); // How many GPUs?
    int device_id = grid3d->iam % deviceCount;
    hipSetDevice(device_id);

    double tRegion[5];
    size_t useableGPUMem = getGPUMemPerProcs(grid3d->comm);
    /**
     *  Memory is divided into two parts data memory and buffer memory
     *  data memory is used for useful data
     *  bufferMemory is used for buffers
     * */
    size_t memReqData = 0;

    /*Memory for XSUP*/
    memReqData += (nsupers + 1) * sizeof(int_t);

    tRegion[0] = SuperLU_timer_();
    size_t totalNzvalSize = 0;
    /*Memory for lapenlPanel Data*/
    for (int_t i = 0; i < CEILING(nsupers, Pc); ++i)
    {
        if (i * Pc + mycol < nsupers && isNodeInMyGrid[i * Pc + mycol] == 1)
        {
            memReqData += lPanelVec[i].totalSize();
            totalNzvalSize += lPanelVec[i].nzvalSize();
        }
    }
    for (int_t i = 0; i < CEILING(nsupers, Pr); ++i)
    {
        if (i * Pr + myrow < nsupers && isNodeInMyGrid[i * Pr + myrow] == 1)
        {
            memReqData += uPanelVec[i].totalSize();
            totalNzvalSize += uPanelVec[i].nzvalSize();
        }
    }
    memReqData += CEILING(nsupers, Pc) * sizeof(lpanelGPU_t);
    memReqData += CEILING(nsupers, Pr) * sizeof(upanelGPU_t);

    memReqData += sizeof(LUstructGPU_t);
    // Per stream data
    // TODO: estimate based on ancestor size
    int_t maxBuffSize = sp_ienv_dist (8, options); //sp_env(8,options);
    A_gpu.gemmBufferSize = SUPERLU_MIN(maxBuffSize, totalNzvalSize);
    size_t dataPerStream = 3 * sizeof(double) * maxLvalCount + 3 * sizeof(double) * maxUvalCount + 2 * sizeof(int_t) * maxLidxCount + 2 * sizeof(int_t) * maxUidxCount + A_gpu.gemmBufferSize * sizeof(double) + ldt * ldt * sizeof(double);
    if (memReqData + 2 * dataPerStream > useableGPUMem)
    {
        printf("Not enough memory on GPU: available = %zu, required for 2 streams =%zu, exiting\n", useableGPUMem, memReqData + 2 * dataPerStream);
        exit(-1);
    }

    tRegion[0] = SuperLU_timer_() - tRegion[0];
    // print the time taken to estimate memory on GPU
    if (grid3d->iam == 0)
    {
        printf("Time taken to estimate memory on GPU: %f\n", tRegion[0]);
    }

    /*Memory for lapenlPanel Data*/
    tRegion[1] = SuperLU_timer_();

    int_t maxNumberOfStream = (useableGPUMem - memReqData) / dataPerStream;

    int numberOfStreams = SUPERLU_MIN(getNumLookAhead(options), maxNumberOfStream);
    numberOfStreams = SUPERLU_MIN(numberOfStreams, MAX_CUDA_STREAMS);
    int rNumberOfStreams;
    MPI_Allreduce(&numberOfStreams, &rNumberOfStreams, 1,
                  MPI_INT, MPI_MIN, grid3d->comm);
    A_gpu.numCudaStreams = rNumberOfStreams;

    if (!grid3d->iam)
        printf("Using %d CUDA LookAhead streams\n", rNumberOfStreams);
    size_t totalMemoryRequired = memReqData + numberOfStreams * dataPerStream;

#if 0
    upanelGPU_t *uPanelVec_GPU = new upanelGPU_t[CEILING(nsupers, Pr)];
    lpanelGPU_t *lPanelVec_GPU = new lpanelGPU_t[CEILING(nsupers, Pc)];
    void *gpuBasePtr, *gpuCurrentPtr;
    hipMalloc(&gpuBasePtr, totalMemoryRequired);
    gpuCurrentPtr = gpuBasePtr;

    A_gpu.xsup = (int_t *)gpuCurrentPtr;
    gpuCurrentPtr = (int_t *)gpuCurrentPtr + (nsupers + 1);
    hipMemcpy(A_gpu.xsup, xsup, (nsupers + 1) * sizeof(int_t), hipMemcpyHostToDevice);

    for (int_t i = 0; i < CEILING(nsupers, Pc); ++i)
    {
        if (i * Pc + mycol < nsupers && isNodeInMyGrid[i * Pc + mycol] == 1)
        {
            lPanelVec_GPU[i] = lPanelVec[i].copyToGPU(gpuCurrentPtr);
            gpuCurrentPtr = (char *)gpuCurrentPtr + lPanelVec[i].totalSize();
        }
    }
    A_gpu.lPanelVec = (lpanelGPU_t *)gpuCurrentPtr;
    gpuCurrentPtr = (char *)gpuCurrentPtr + CEILING(nsupers, Pc) * sizeof(lpanelGPU_t);
    hipMemcpy(A_gpu.lPanelVec, lPanelVec_GPU,
               CEILING(nsupers, Pc) * sizeof(lpanelGPU_t), hipMemcpyHostToDevice);

    for (int_t i = 0; i < CEILING(nsupers, Pr); ++i)
    {
        if (i * Pr + myrow < nsupers && isNodeInMyGrid[i * Pr + myrow] == 1)
        {
            uPanelVec_GPU[i] = uPanelVec[i].copyToGPU(gpuCurrentPtr);
            gpuCurrentPtr = (char *)gpuCurrentPtr + uPanelVec[i].totalSize();
        }
    }
    A_gpu.uPanelVec = (upanelGPU_t *)gpuCurrentPtr;
    gpuCurrentPtr = (char *)gpuCurrentPtr + CEILING(nsupers, Pr) * sizeof(upanelGPU_t);
    hipMemcpy(A_gpu.uPanelVec, uPanelVec_GPU,
               CEILING(nsupers, Pr) * sizeof(upanelGPU_t), hipMemcpyHostToDevice);

    for (int stream = 0; stream < A_gpu.numCudaStreams; stream++)
    {

        hipStreamCreate(&A_gpu.cuStreams[stream]);
        hipblasCreate(&A_gpu.cuHandles[stream]);
        A_gpu.LvalRecvBufs[stream] = (double *)gpuCurrentPtr;
        gpuCurrentPtr = (double *)gpuCurrentPtr + maxLvalCount;
        A_gpu.UvalRecvBufs[stream] = (double *)gpuCurrentPtr;
        gpuCurrentPtr = (double *)gpuCurrentPtr + maxUvalCount;
        A_gpu.LidxRecvBufs[stream] = (int_t *)gpuCurrentPtr;
        gpuCurrentPtr = (int_t *)gpuCurrentPtr + maxLidxCount;
        A_gpu.UidxRecvBufs[stream] = (int_t *)gpuCurrentPtr;
        gpuCurrentPtr = (int_t *)gpuCurrentPtr + maxUidxCount;

        A_gpu.gpuGemmBuffs[stream] = (double *)gpuCurrentPtr;
        gpuCurrentPtr = (double *)gpuCurrentPtr + A_gpu.gemmBufferSize;
        A_gpu.dFBufs[stream] = (double *)gpuCurrentPtr;
        gpuCurrentPtr = (double *)gpuCurrentPtr + ldt * ldt;

        /*lookAhead buffers and stream*/
        hipblasCreate(&A_gpu.lookAheadLHandle[stream]);
        hipStreamCreate(&A_gpu.lookAheadLStream[stream]);
        A_gpu.lookAheadLGemmBuffer[stream] = (double *)gpuCurrentPtr;
        gpuCurrentPtr = (double *)gpuCurrentPtr + maxLvalCount;
        hipblasCreate(&A_gpu.lookAheadUHandle[stream]);
        hipStreamCreate(&A_gpu.lookAheadUStream[stream]);
        A_gpu.lookAheadUGemmBuffer[stream] = (double *)gpuCurrentPtr;
        gpuCurrentPtr = (double *)gpuCurrentPtr + maxUvalCount;
    }
    // cudaCheckError();
    // allocate
    dA_gpu = (LUstructGPU_t *)gpuCurrentPtr;

    hipMemcpy(dA_gpu, &A_gpu, sizeof(LUstructGPU_t), hipMemcpyHostToDevice);
    gpuCurrentPtr = (LUstructGPU_t *)gpuCurrentPtr + 1;

#else
    hipMalloc(&A_gpu.xsup, (nsupers + 1) * sizeof(int_t));
    hipMemcpy(A_gpu.xsup, xsup, (nsupers + 1) * sizeof(int_t), hipMemcpyHostToDevice);

    double tLsend, tUsend;

    tLsend = SuperLU_timer_();
    upanelGPU_t *uPanelVec_GPU = copyUpanelsToGPU();
    tLsend = SuperLU_timer_() - tLsend;
    tUsend = SuperLU_timer_();
    lpanelGPU_t *lPanelVec_GPU = copyLpanelsToGPU();
    tUsend = SuperLU_timer_() - tUsend;

    tRegion[1] = SuperLU_timer_() - tRegion[1];
    printf("TRegion L,U send: \t %g\n", tRegion[1]);
    printf("Time to send Lpanel=%g  and U panels =%g \n", tLsend, tUsend);

    hipMalloc(&A_gpu.lPanelVec, CEILING(nsupers, Pc) * sizeof(lpanelGPU_t));
    hipMemcpy(A_gpu.lPanelVec, lPanelVec_GPU,
               CEILING(nsupers, Pc) * sizeof(lpanelGPU_t), hipMemcpyHostToDevice);
    hipMalloc(&A_gpu.uPanelVec, CEILING(nsupers, Pr) * sizeof(upanelGPU_t));
    hipMemcpy(A_gpu.uPanelVec, uPanelVec_GPU,
               CEILING(nsupers, Pr) * sizeof(upanelGPU_t), hipMemcpyHostToDevice);

    tRegion[2] = SuperLU_timer_();
    int dfactBufSize = 0;
    // TODO: does it work with NULL pointer?
    hipsolverHandle_t cusolverH = NULL;
    hipsolverDnCreate(&cusolverH);
    
    hipsolverDnDgetrf_bufferSize(cusolverH, ldt, ldt, NULL, ldt, &dfactBufSize);
    printf("Size of dfactBuf is %d\n", dfactBufSize);
    tRegion[2] = SuperLU_timer_() - tRegion[2];
    printf("TRegion dfactBuf: \t %g\n", tRegion[2]);
    

    tRegion[3] = SuperLU_timer_();

    double tcuMalloc=SuperLU_timer_();
    for (int stream = 0; stream < A_gpu.numCudaStreams; stream++)
    {
        hipMalloc(&A_gpu.LvalRecvBufs[stream], sizeof(double) * maxLvalCount);
        hipMalloc(&A_gpu.UvalRecvBufs[stream], sizeof(double) * maxUvalCount);
        hipMalloc(&A_gpu.LidxRecvBufs[stream], sizeof(int_t) * maxLidxCount);
        hipMalloc(&A_gpu.UidxRecvBufs[stream], sizeof(int_t) * maxUidxCount);
        // allocate the space for diagonal factor on GPU
        hipMalloc(&A_gpu.diagFactWork[stream], sizeof(double) * dfactBufSize);
        hipMalloc(&A_gpu.diagFactInfo[stream], sizeof(int));

        hipMalloc(&A_gpu.gpuGemmBuffs[stream], A_gpu.gemmBufferSize * sizeof(double));
        /*lookAhead buffers and stream*/
        hipMalloc(&A_gpu.lookAheadLGemmBuffer[stream], sizeof(double) * maxLvalCount);

        hipMalloc(&A_gpu.lookAheadUGemmBuffer[stream], sizeof(double) * maxUvalCount);

        hipMalloc(&A_gpu.dFBufs[stream], ldt * ldt * sizeof(double));
    }
    tcuMalloc = SuperLU_timer_() - tcuMalloc;
    printf("Time to allocate GPU memory: %g\n", tcuMalloc);

    double tcuStream=SuperLU_timer_();
    
    for (int stream = 0; stream < A_gpu.numCudaStreams; stream++)
    {
        hipblasCreate(&A_gpu.cuHandles[stream]);
        hipsolverDnCreate(&A_gpu.cuSolveHandles[stream]);
    }
    tcuStream = SuperLU_timer_() - tcuStream;
    printf("Time to create cublas streams: %g\n", tcuStream);

    double tcuStreamCreate=SuperLU_timer_();
    for (int stream = 0; stream < A_gpu.numCudaStreams; stream++)
    {

        hipStreamCreate(&A_gpu.cuStreams[stream]);
        hipblasCreate(&A_gpu.cuHandles[stream]);
        /*lookAhead buffers and stream*/
        hipblasCreate(&A_gpu.lookAheadLHandle[stream]);
        hipStreamCreate(&A_gpu.lookAheadLStream[stream]);
        hipblasCreate(&A_gpu.lookAheadUHandle[stream]);
        hipStreamCreate(&A_gpu.lookAheadUStream[stream]);
    }
    tcuStreamCreate = SuperLU_timer_() - tcuStreamCreate;
    printf("Time to create CUDA streams: %g\n", tcuStreamCreate);

    tRegion[3] = SuperLU_timer_() - tRegion[3];
    printf("TRegion stream: \t %g\n", tRegion[3]);
    // allocate
    hipMalloc(&dA_gpu, sizeof(LUstructGPU_t));
    hipMemcpy(dA_gpu, &A_gpu, sizeof(LUstructGPU_t), hipMemcpyHostToDevice);

#endif
    // cudaCheckError();
    return 0;
}

int_t LUstruct_v100::copyLUGPUtoHost()
{

    for (int_t i = 0; i < CEILING(nsupers, Pc); ++i)
        if (i * Pc + mycol < nsupers && isNodeInMyGrid[i * Pc + mycol] == 1)
            lPanelVec[i].copyFromGPU();

    for (int_t i = 0; i < CEILING(nsupers, Pr); ++i)
        if (i * Pr + myrow < nsupers && isNodeInMyGrid[i * Pr + myrow] == 1)
            uPanelVec[i].copyFromGPU();
    return 0;
}

int_t LUstruct_v100::copyLUHosttoGPU()
{
    for (int_t i = 0; i < CEILING(nsupers, Pc); ++i)
        if (i * Pc + mycol < nsupers && isNodeInMyGrid[i * Pc + mycol] == 1)
            lPanelVec[i].copyToGPU();

    for (int_t i = 0; i < CEILING(nsupers, Pr); ++i)
        if (i * Pr + myrow < nsupers && isNodeInMyGrid[i * Pr + myrow] == 1)
            uPanelVec[i].copyToGPU();
    return 0;
}

int_t LUstruct_v100::checkGPU()
{

    for (int_t i = 0; i < CEILING(nsupers, Pc); ++i)
        lPanelVec[i].checkGPU();

    for (int_t i = 0; i < CEILING(nsupers, Pr); ++i)
        uPanelVec[i].checkGPU();

    std::cout << "Checking LU struct completed succesfully"
              << "\n";
    return 0;
}

/**
 * @brief Pack non-zero values into a vector.
 *
 * @param spNzvalArray The array of non-zero values.
 * @param nzvalSize The size of the array of non-zero values.
 * @param valOffset The offset of the non-zero values.
 * @param packedNzvals The vector to store the non-zero values.
 * @param packedNzvalsIndices The vector to store the indices of the non-zero values.
 */
void packNzvals(std::vector<double> &packedNzvals, std::vector<int_t> &packedNzvalsIndices,
                double *spNzvalArray, int_t nzvalSize, int_t valOffset)
{
    for (int k = 0; k < nzvalSize; k++)
    {
        if (spNzvalArray[k] != 0)
        {
            packedNzvals.push_back(spNzvalArray[k]);
            packedNzvalsIndices.push_back(valOffset + k);
        }
    }
}

const int AVOID_CPU_NZVAL = 1;
lpanelGPU_t *LUstruct_v100::copyLpanelsToGPU()
{
    lpanelGPU_t *lPanelVec_GPU = new lpanelGPU_t[CEILING(nsupers, Pc)];

    // TODO: set gpuLvalSize, gpuLidxSize
    gpuLvalSize = 0;
    gpuLidxSize = 0;
    for (int_t i = 0; i < CEILING(nsupers, Pc); ++i)
    {
        if (i * Pc + mycol < nsupers && isNodeInMyGrid[i * Pc + mycol] == 1)
        {
            gpuLvalSize += sizeof(double) * lPanelVec[i].nzvalSize();
            gpuLidxSize += sizeof(int_t) * lPanelVec[i].indexSize();
        }
    }

    double *valBuffer = (double *)SUPERLU_MALLOC(gpuLvalSize);
    int_t *idxBuffer = (int_t *)SUPERLU_MALLOC(gpuLidxSize);

    // allocate memory buffer on GPU
    hipMalloc(&gpuLvalBasePtr, gpuLvalSize);
    hipMalloc(&gpuLidxBasePtr, gpuLidxSize);

    size_t valOffset = 0;
    size_t idxOffset = 0;
    double tCopyToCPU = SuperLU_timer_();

    std::vector<double> packedNzvals;
    std::vector<int_t> packedNzvalsIndices;

    // do a memcpy to CPU buffer
    for (int_t i = 0; i < CEILING(nsupers, Pc); ++i)
    {
        if (i * Pc + mycol < nsupers && isNodeInMyGrid[i * Pc + mycol] == 1)
        {
            if (lPanelVec[i].isEmpty())
            {
                lpanelGPU_t ithLpanel(NULL, NULL);
                lPanelVec[i].gpuPanel = ithLpanel;
                lPanelVec_GPU[i] = ithLpanel;
            }
            else
            {
                lpanelGPU_t ithLpanel(&gpuLidxBasePtr[idxOffset], &gpuLvalBasePtr[valOffset]);
                lPanelVec[i].gpuPanel = ithLpanel;
                lPanelVec_GPU[i] = ithLpanel;
                if (AVOID_CPU_NZVAL)
                    packNzvals(packedNzvals, packedNzvalsIndices, lPanelVec[i].val, lPanelVec[i].nzvalSize(), valOffset);
                else
                    memcpy(&valBuffer[valOffset], lPanelVec[i].val, sizeof(double) * lPanelVec[i].nzvalSize());

                memcpy(&idxBuffer[idxOffset], lPanelVec[i].index, sizeof(int_t) * lPanelVec[i].indexSize());

                valOffset += lPanelVec[i].nzvalSize();
                idxOffset += lPanelVec[i].indexSize();
            }
        }
    }
    tCopyToCPU = SuperLU_timer_() - tCopyToCPU;
    std::cout << "Time to copy-L to CPU: " << tCopyToCPU << "\n";
    // do a hipMemcpy to GPU
    double tLsend = SuperLU_timer_();
    if (AVOID_CPU_NZVAL)
        copyToGPU(gpuLvalBasePtr, packedNzvals, packedNzvalsIndices);
    else
    {
#if 0
            hipMemcpy(gpuLvalBasePtr, valBuffer, gpuLvalSize, hipMemcpyHostToDevice);
#else
        copyToGPU_Sparse(gpuLvalBasePtr, valBuffer, gpuLvalSize);
#endif
    }
    // find
    hipMemcpy(gpuLidxBasePtr, idxBuffer, gpuLidxSize, hipMemcpyHostToDevice);
    tLsend = SuperLU_timer_() - tLsend;
    printf("hipMemcpy time L =%g \n", tLsend);

    SUPERLU_FREE(valBuffer);
    SUPERLU_FREE(idxBuffer);
    return lPanelVec_GPU;
}

upanelGPU_t *LUstruct_v100::copyUpanelsToGPU()
{
    upanelGPU_t *uPanelVec_GPU = new upanelGPU_t[CEILING(nsupers, Pr)];

    gpuUvalSize = 0;
    gpuUidxSize = 0;
    for (int_t i = 0; i < CEILING(nsupers, Pr); ++i)
    {
        if (i * Pr + myrow < nsupers && isNodeInMyGrid[i * Pr + myrow] == 1)
        {
            gpuUvalSize += sizeof(double) * uPanelVec[i].nzvalSize();
            gpuUidxSize += sizeof(int_t) * uPanelVec[i].indexSize();
        }
    }

    // TODO: set gpuUvalSize, gpuUidxSize

    // allocate memory buffer on GPU
    hipMalloc(&gpuUvalBasePtr, gpuUvalSize);
    hipMalloc(&gpuUidxBasePtr, gpuUidxSize);

    size_t valOffset = 0;
    size_t idxOffset = 0;

    double tCopyToCPU = SuperLU_timer_();
    for (int_t i = 0; i < CEILING(nsupers, Pr); ++i)
    {
        if (i * Pr + myrow < nsupers && isNodeInMyGrid[i * Pr + myrow] == 1)
        {
            if (uPanelVec[i].isEmpty())
            {
                upanelGPU_t ithupanel(NULL, NULL);
                uPanelVec[i].gpuPanel = ithupanel;
                uPanelVec_GPU[i] = ithupanel;
            }
        }
    }

    int_t *idxBuffer = (int_t *)SUPERLU_MALLOC(gpuUidxSize);

    if (AVOID_CPU_NZVAL)
    {
        printf("AVOID_CPU_NZVAL is set\n");
        std::vector<double> packedNzvals;
        std::vector<int_t> packedNzvalsIndices;
        for (int_t i = 0; i < CEILING(nsupers, Pr); ++i)
        {
            if (i * Pr + myrow < nsupers && isNodeInMyGrid[i * Pr + myrow] == 1)
            {
                if (!uPanelVec[i].isEmpty())
                {

                    upanelGPU_t ithupanel(&gpuUidxBasePtr[idxOffset], &gpuUvalBasePtr[valOffset]);
                    uPanelVec[i].gpuPanel = ithupanel;
                    uPanelVec_GPU[i] = ithupanel;
                    packNzvals(packedNzvals, packedNzvalsIndices, uPanelVec[i].val,
                               uPanelVec[i].nzvalSize(), valOffset);
                    memcpy(&idxBuffer[idxOffset], uPanelVec[i].index, sizeof(int_t) * uPanelVec[i].indexSize());

                    valOffset += uPanelVec[i].nzvalSize();
                    idxOffset += uPanelVec[i].indexSize();
                }
            }
        }
        tCopyToCPU = SuperLU_timer_() - tCopyToCPU;
        printf("copyU to CPU-buff time = %g\n", tCopyToCPU);

        // do a hipMemcpy to GPU
        double tLsend = SuperLU_timer_();
        copyToGPU(gpuUvalBasePtr, packedNzvals, packedNzvalsIndices);
        hipMemcpy(gpuUidxBasePtr, idxBuffer, gpuUidxSize, hipMemcpyHostToDevice);
        tLsend = SuperLU_timer_() - tLsend;
        printf("hipMemcpy time U =%g \n", tLsend);
        // SUPERLU_FREE(valBuffer);
    }
    else
    {
        // do a memcpy to CPU buffer
        double *valBuffer = (double *)SUPERLU_MALLOC(gpuUvalSize);

        for (int_t i = 0; i < CEILING(nsupers, Pr); ++i)
        {
            if (i * Pr + myrow < nsupers && isNodeInMyGrid[i * Pr + myrow] == 1)
            {
                if (!uPanelVec[i].isEmpty())
                {

                    upanelGPU_t ithupanel(&gpuUidxBasePtr[idxOffset], &gpuUvalBasePtr[valOffset]);
                    uPanelVec[i].gpuPanel = ithupanel;
                    uPanelVec_GPU[i] = ithupanel;
                    memcpy(&valBuffer[valOffset], uPanelVec[i].val, sizeof(double) * uPanelVec[i].nzvalSize());
                    memcpy(&idxBuffer[idxOffset], uPanelVec[i].index, sizeof(int_t) * uPanelVec[i].indexSize());

                    valOffset += uPanelVec[i].nzvalSize();
                    idxOffset += uPanelVec[i].indexSize();
                }
            }
        }
        tCopyToCPU = SuperLU_timer_() - tCopyToCPU;
        printf("copyU to CPU-buff time = %g\n", tCopyToCPU);

        // do a hipMemcpy to GPU
        double tLsend = SuperLU_timer_();
        const int USE_GPU_COPY = 1;
        if (USE_GPU_COPY)
            hipMemcpy(gpuUvalBasePtr, valBuffer, gpuUvalSize, hipMemcpyHostToDevice);
        else
            copyToGPU_Sparse(gpuUvalBasePtr, valBuffer, gpuUvalSize);

        hipMemcpy(gpuUidxBasePtr, idxBuffer, gpuUidxSize, hipMemcpyHostToDevice);
        tLsend = SuperLU_timer_() - tLsend;
        printf("hipMemcpy time U =%g \n", tLsend);
        SUPERLU_FREE(valBuffer);
    }
    SUPERLU_FREE(idxBuffer);
    return uPanelVec_GPU;
}